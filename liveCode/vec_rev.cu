#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

hipError_t addWithCuda(unsigned int size, const float *a, const float *b, float *c);

__global__ void addKernel(int N, const float *a, const float *b, float *c)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < N) {
		c[i] = a[i] + b[i];
	}
}

__global__ void revKernel(int N, float *a, float *b)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < N) {
		b[N - i - 1] = a[i];
	}
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t revWithCuda(unsigned int size, float *a, float *b)
{
	float *dev_a = 0;
	float *dev_b = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int nt = 10;
	int nb = (size + nt - 1) / nt;
	revKernel << <nb, nt >> >(size, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(b, dev_b, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

void addVectors(int N, float *a, float *b, float *c) {
	int i;

	for (i = 0; i < N; i++) {
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int N = 100;
	int i;
	float *a = (float*)malloc(N * sizeof(float));
	float *b = (float*)malloc(N * sizeof(float));
	float *c = (float*)malloc(N * sizeof(float));

	for (i = 0; i < N; i++) {
		a[i] = i;
	}

	//addVectors(N, a, b, c);
	//addWithCuda(N, a, b, c);
	revWithCuda(N, a, b);

	for (i = 0; i < 5; i++) {
		printf("b[%d] = %f\n", i, b[i]);
	}

	free(a);
	free(b);
	free(c);
    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(unsigned int size, const float *a, const float *b, float *c)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	int nt = 10;
	int nb = (size + nt - 1) / nt;
    addKernel<<<nb, nt>>>(size, dev_a, dev_b, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
